#include "hip/hip_runtime.h"
/* Brick Sort - parallel Bubble sort 
   Implemented by: Ksenia Burova

   March 30th, 2018 */

#include <thrust/swap.h>
#include "../helper.h"
#include <hip/hip_runtime.h>
#include <chrono>

using namespace std;

__global__ void SortEven(int *d_arr, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i%2 == 0 && i < size -1) {
        if (d_arr[i] > d_arr[i+1])
            thrust::swap(d_arr[i], d_arr[i+1]);
    }
}

__global__ void SortOdd(int *d_arr, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i%2 == 1 && i < size -1) {
        if (d_arr[i] > d_arr[i+1])
            thrust::swap(d_arr[i], d_arr[i+1]);
    }
}

void sort(int *d_arr, int size, int num_blocks, int num_threads) {
    int i;
    for (i = 0; i <= size/2; i++) {
        SortEven <<< num_blocks, num_threads >>>(d_arr, size);
        SortOdd <<< num_blocks, num_threads >>>(d_arr,size);
    }
}

int main(int argc, char **argv) {

    int *h_arr, *d_arr;
    int size, step, max;
    int num_blocks, num_threads;
    double diff;
    chrono::high_resolution_clock::time_point start, stop;

    step = 1000;
    max = 10000000;

    h_arr = (int*)calloc(0,sizeof(int));
    
    for (size = 1000; size < max; size = 2*step, step = 1.1*step) {
        num_blocks = ceil(size/1024.0);
        num_threads = (num_blocks > 1) ? 1024 : size;
        
        h_arr = (int*)realloc(h_arr, sizeof(int) * size);
        populate_array(h_arr, size);
        
        hipMalloc(&d_arr, sizeof(int)*size);
        hipMemcpy(d_arr, h_arr, sizeof(int) * size, hipMemcpyHostToDevice);
        
        start = chrono::high_resolution_clock::now();
        sort(d_arr, size, num_blocks, num_threads);
        stop = chrono::high_resolution_clock::now();
        diff = chrono::duration <double, milli> (stop-start).count();
        
        hipMemcpy(h_arr, d_arr, sizeof(int) * size, hipMemcpyDeviceToHost);
        hipFree(d_arr);
        
        printf("%d%c%0.6f\n", size,'\t',diff);
        fflush(stdout);
    }

    free(h_arr);

    return 0;
}
